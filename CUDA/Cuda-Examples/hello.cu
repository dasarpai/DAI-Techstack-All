// hello.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
    printf("Hello, CUDA World from GPU!\n");
}

int main() {
    // Launch the kernel with one block and one thread
    hello_cuda<<<1, 1>>>();

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    printf("Hello, CUDA World from CPU!\n");
    return 0;
}

